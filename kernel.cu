/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
    

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
	//float *d_A,*d_B,*d_C;
	//int size = n*sizeof(float);

	/*cudaError_t err =*/// cudaMalloc(&d_A,size);
	/*if (err != cudaSuccess) {
		 printf(“%s in %s at line %d\n”, __LINE__); exit(EXIT_FAILURE);
	}
	err =*/// cudaMalloc(&d_B,size);
	/*if (err != cudaSuccess) {
		 printf(“%s in %s at line %d\n”, __LINE__); exit(EXIT_FAILURE);
	}	
//	err = */    // cudaMalloc(&d_C,size);
	/*if (err != cudaSuccess) {
		 printf(“%s in %s at line %d\n”, __LINE__); exit(EXIT_FAILURE);
	}*/
//	cudaMemcpy(d_A, A, size, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_B, B, size, cudaMemcpyHostToDevice);
	
//	int i = 0;
//	for(i=0;i<n;i++){
//		d_C[i] = d_A[i] + d_B[i];
//	}






	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n) C[i] = A[i] + B[i];	
}


void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 256; //changed from 512 

    //INSERT CODE HERE
		

	dim3 DimGrid((n-1)/BLOCK_SIZE + 1, 1,1);
	dim3 DimBlock(BLOCK_SIZE,1,1);	
	
	

	
	
	
		
	VecAdd<<<DimGrid,DimBlock>>>(n,A,B,C);

	
	
		

	//dim3 DimGrid(ceil(n/256),1,1);
	//dim3 DimBlock(256,1,1);
	//vecAddKernel<<<DimGrid,DimBlock>>(d_A,d_B,d_C,n);

}

